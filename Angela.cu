// ----------------------------------------------------------------
// Sudoku -- Puzzle Solver on GPU using CUDA
// ----------------------------------------------------------------

/**
 * @file
 * sudoku.cu
 *
 * @brief main sudoku file to init and execute
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// includes, utilities
#include "util/error_utils.cuh"
#include "util/io_utils.cuh"
#include "data.cuh"

// includes, kernels
//#include "beecolony.cuh"
#include "AngelaKernels.cuh"

//#include "bfsKernel.cuh"

void KernelManager(int n, Square * h_unsolved, bool o_graphics) {

  /* CUDA event setup */
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  /* Memory Allocations */
  int memsize = sizeof(Square) * n * n;

  Square * d_unsolved;
  ERROR_CHECK( hipMalloc((void**) &d_unsolved, memsize) );
  /* IMPORTANT: PLEASE ADD THIS IN YOUR KERNEL MANAGER FUNCTION */
  /*ERROR_CHECK( hipMemcpy(d_unsolved, h_unsolved, memsize,
                          hipMemcpyHostToDevice) );*/
 /* IMPORTANT: END! */


  Square * d_solved;
  ERROR_CHECK( hipMalloc((void**) &d_solved, memsize) );

//  int* d_points;
//  ERROR_CHECK( hipMalloc((void**) &d_points, sizeof(int)) );


  float elapsedTime;
  hipEventRecord(start, 0);
//  ArtificialBeeColony (h_unsolved, d_unsolved, d_solved, n);
  AngelaKernels(h_unsolved, d_unsolved, d_solved, n);
//  bfsKernel( h_unsolved, d_unsolved, d_solved, n);



  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);

  Square * h_solved = (Square *) malloc(memsize);
  ERROR_CHECK( hipMemcpy(h_solved, d_unsolved, memsize,
                          hipMemcpyDeviceToHost) );

  /* Destroy CUDA event */
  hipEventDestroy(start);
  hipEventDestroy(stop);

  // TODO: Terminal Output will go here.
  const char * alg = "-ang";

    const char * finished = "/********** Angela's (C) **********/";
    output(finished, alg, n, false, h_solved);

  const char* statistics = "/******* Statistics (Begin) ********/";
  printf("%s\n", statistics);
  printf("Elapsed Time: %f (ms)\n", elapsedTime);
  const char* statistics_end = "/******** Statistics (End) *********/";
  printf("%s\n", statistics_end);

  /* Free Memory Allocations */
  free(h_unsolved);
  ERROR_CHECK( hipFree(d_unsolved) );
  ERROR_CHECK( hipFree(d_solved) );
}

int main(int argc, char** argv) {

    /* Gets arguments from command line and puzzle from a file */
    CommandLineArgs * build = new CommandLineArgs;
    input(argc, argv, build);
    KernelManager((*build).size, (*build).Puzzle, (*build).graphics);

}
