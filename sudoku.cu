// ----------------------------------------------------------------
// Sudoku -- Puzzle Solver on GPU using CUDA
// ----------------------------------------------------------------

/**
 * @file
 * sudoku.cu
 *
 * @brief main sudoku file to init and execute
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// includes, utilities
#include "util/error_utils.cuh"
#include "util/io_utils.cuh"
#include "data.cuh"

// includes, kernels
// #include "kernels.cuh"
#include "beecolony.cuh"

void KernelManager(int n, Square * h_unsolved, bool o_graphics) {

  /* CUDA event setup */
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  /* Memory Allocations */
  int memsize = sizeof(Square) * n * n;

  Square * d_unsolved;
  ERROR_CHECK( hipMalloc((void**) &d_unsolved, memsize) );
  ERROR_CHECK( hipMemcpy(d_unsolved, h_unsolved, memsize,
                          hipMemcpyHostToDevice) );

  Square * d_solved;
  ERROR_CHECK( hipMalloc((void**) &d_solved, memsize) );

  float elapsedTime;
  hipEventRecord(start, 0);

  ArtificialBeeColony (d_unsolved, d_solved, n);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);

  Square * h_solved = (Square *) malloc(memsize);
  ERROR_CHECK( hipMemcpy(h_solved, d_solved, memsize,
                          hipMemcpyDeviceToHost) );

  /* Destroy CUDA event */
  hipEventDestroy(start);
  hipEventDestroy(stop);

  // TODO: Terminal Output will go here.
  const char * finished = "/********** Output Puzzle **********/";
  output(finished, n, false, h_solved);

  /* Free Memory Allocations */
  free(h_unsolved);
  ERROR_CHECK( hipFree(d_unsolved) );
  ERROR_CHECK( hipFree(d_solved) );
}

int main(int argc, char** argv) {

    /* Gets arguments from command line and puzzle from a file */
    CommandLineArgs * build = new CommandLineArgs;
    input(argc, argv, build);
    KernelManager((*build).size, (*build).Puzzle, (*build).graphics);

}
