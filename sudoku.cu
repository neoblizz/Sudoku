#include "hip/hip_runtime.h"
// ----------------------------------------------------------------
// Sudoku -- Puzzle Solver on GPU using CUDA
// ----------------------------------------------------------------

/**
 * @file
 * sudoku.cu
 *
 * @brief main sudoku file to init and execute
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// includes, utilities
#include "util/error_utils.cuh"
#include "util/io_utils.cuh"
#include "data.cuh"

// includes, kernels
#include "beecolony.cuh"
#include "humanlogic.cuh"
#include "breadthfirstsearch.cuh"

void print (int n, Square * h_solved, const char * alg, float elapsedTime) {

  if (!strcmp(alg, "-bee")) {
    CSRtoCSC(h_solved, n);
    const char * finished = "/********** Bee Colony (C) **********/";
    output(finished, alg, n, false, h_solved);
  } else if (!strcmp(alg, "-log")) {
    const char * finished = "/********** Human-Logic(C) **********/";
    output(finished, alg, n, false, h_solved);
  } else if (!strcmp(alg, "-bfs")) {
    const char * finished = "/************ BFS-DFS (C) ***********/";
    output(finished, alg, n, false, h_solved);
  }

  const char* statistics = "/******* Statistics (Begin) ********/";
  printf("%s\n", statistics);
  printf("Elapsed Time: %f (ms)\n", elapsedTime);
  const char* statistics_end = "/******** Statistics (End) *********/";
  printf("%s\n", statistics_end);
}

void release (Square * h_unsolved, Square * d_unsolved, Square * d_solved) {
  /* Free Memory Allocations */
  free(h_unsolved);
  ERROR_CHECK( hipFree(d_unsolved) );
  ERROR_CHECK( hipFree(d_solved) );
}

void KernelManager(int n, Square * h_unsolved, bool o_graphics) {

  /* CUDA event setup */
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  /* Memory Allocations */
  int memsize = sizeof(Square) * n * n;

  Square * d_unsolved;
  ERROR_CHECK( hipMalloc((void**) &d_unsolved, memsize) );
  /* IMPORTANT: PLEASE ADD THIS IN YOUR KERNEL MANAGER FUNCTION */
  /*ERROR_CHECK( hipMemcpy(d_unsolved, h_unsolved, memsize,
                          hipMemcpyHostToDevice) );*/
 /* IMPORTANT: END! */


  Square * d_solved;
  ERROR_CHECK( hipMalloc((void**) &d_solved, memsize) );

  /* ARTIFICIAL BEE COLONY ALGORITHM */
  float elapsedTime;
  hipEventRecord(start, 0);
  ArtificialBeeColony (h_unsolved, d_unsolved, d_solved, n);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);

  Square * h_solved = (Square *) malloc(memsize);
  ERROR_CHECK( hipMemcpy(h_solved, d_solved, memsize,
                          hipMemcpyDeviceToHost) );

  const char * bee = "-bee";
  print(n, h_solved, bee, elapsedTime);

  /* HUMAN LOGIC BASED ALGORITHM */
  hipEventRecord(start, 0);
  HumanLogic (h_unsolved, d_unsolved, d_solved, n);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);

  ERROR_CHECK( hipMemcpy(h_solved, d_solved, memsize,
                          hipMemcpyDeviceToHost) );

  const char * logic = "-log";
  print(n, h_solved, logic, elapsedTime);

  /* TREE BASED ALGORITHM */
  hipEventRecord(start, 0);
  BreadthFirstSearch (h_unsolved, d_unsolved, d_solved, n);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);

  ERROR_CHECK( hipMemcpy(h_solved, d_solved, memsize,
                          hipMemcpyDeviceToHost) );

  const char * tree = "-bfs";
  print(n, h_solved, tree, elapsedTime);

  /* Destroy CUDA event */
  hipEventDestroy(start);
  hipEventDestroy(stop);

  /* Free Memory */
  release(h_unsolved, d_unsolved, d_solved);

}

int main(int argc, char** argv) {

    /* Gets arguments from command line and puzzle from a file */
    CommandLineArgs * build = new CommandLineArgs;
    input(argc, argv, build);
    KernelManager((*build).size, (*build).Puzzle, (*build).graphics);

}
